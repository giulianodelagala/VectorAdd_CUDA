#include <iostream>
#include <time.h>
#include <math.h>

//#include <cuda.h>
#include <hip/hip_runtime.h>


using std::cout; using std::cin;

//void ImpError(cudaError_t err);

void ImpError(hipError_t err)
{
	cout << hipGetErrorString(err); // << " en " << __FILE__ << __LINE__;
	//exit(EXIT_FAILURE);
}


__global__
void vecAddKernel(float* A, float* B, float* C, int n)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < n)
		C[i] = A[i] + B[i];
}


void vecAdd(float* A, float* B, float* C, int n)
{
	int size = n * sizeof(float);
	float* d_A, * d_B, * d_C;

	hipError_t err = hipSuccess;
	
	err = hipMalloc((void**)& d_A, size);

	if (err != hipSuccess)
	{
		cout << "d_A";
		ImpError(err);
	}
		

	err = hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
		ImpError(err);

	err = hipMalloc((void**)& d_B, size);

	if (err != hipSuccess)
		ImpError(err);

	err = hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
		ImpError(err);

	err = hipMalloc((void**)& d_C, size);

	if (err != hipSuccess)
		ImpError(err);

	//<<#bloques,#threads por bloques>>
	vecAddKernel<<<ceil(n / 512.0), 512>>>(d_A, d_B, d_C, n);

	err = hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

	if (err != hipSuccess)
		ImpError(err);
	
	hipFree(d_A); hipFree(d_B); hipFree(d_C);
}

void Imprimir(float* A, int n)
{
	for (int i = 0; i < n; ++i)
		if (i<n) cout << A[i] << " ";
	cout << "\n";
}

void GenVector(float* A, int n)
{
	
	for (int i = 0; i < n; ++i)
		A[i] = static_cast <float> (rand()) / (static_cast <float> (RAND_MAX / n));
}


int main(int argc, char** argv)
{
	int array_size = 10;
	
	float* A, * B, * C;
	srand(time(NULL));
	/*
	if (argc == 2)
	{
		array_size = strtof(argv[1], NULL);
	}
	else
		cout << "Ingrese array_size"; cin >> array_size;
	*/

	A = new float[array_size];
	B = new float[array_size];
	C = new float[array_size];

	GenVector(A, array_size);
	GenVector(B, array_size);
	
	vecAdd(A, B, C, array_size);

	Imprimir(A, array_size);
	Imprimir(B, array_size);
	Imprimir(C, array_size);

	//cudaDeviceSynchronize();

	return 0;
}